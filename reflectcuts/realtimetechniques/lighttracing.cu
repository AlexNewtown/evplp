#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_device.h>

#include <hiprand/hiprand_kernel.h>

#include "rtmath.cuh"
#include "rtlightsource.cuh"
#include "rtmaterial.cuh"
#include "rtcomphoton/rtphotonrecord.h"

///// shared info /////

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDimension, rtLaunchDim, );

rtDeclareVariable(uint, rngSeed, , );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(uint, numLightPaths, , );
rtDeclareVariable(uint, numPhotonsPerLightPath, , );
rtDeclareVariable(uint, numVplLightPaths, , );

///// Light Trace info /////

rtBuffer<RtPhotonRecord, 1> photons;
rtBuffer<RtPhotonInfo, 1> photonInfo;

//rtDeclareVariable(uint, numMaxPhotonsPerLightPath, , );

///// VPL splat /////

rtBuffer<float4, 2> outputBuffer;

rtTextureSampler<float4, 2> deferredPositionTexture;
rtTextureSampler<float4, 2> deferredNormalTexture;
rtTextureSampler<float4, 2> deferredDiffuseTexture;
rtTextureSampler<float4, 2> deferredPhongReflectanceTexture;
rtTextureSampler<float4, 2> deferredPhongExpostureTexture;

rtDeclareVariable(float3, cameraPosition, , );
rtDeclareVariable(uint, doAccumulate, , );
rtDeclareVariable(float, pdfMc, , );
rtDeclareVariable(float, radius, , );
rtDeclareVariable(uint, misMode, , );
rtDeclareVariable(float, clampingValue, , );

rtDeclareVariable(float, vslRadius, , );
rtDeclareVariable(float, vslInvPiRadius2, , );

RT_PROGRAM void insertPhotons(
	const unsigned int pmIndex,
	const unsigned int numBounce,
	const float3 & position)
{
	RtPhotonRecord & rec = photons[pmIndex + numBounce];
	rec.mPosition = position;
}

RT_PROGRAM void exception()
{
	rtPrintExceptionDetails();
}

// ray type 0
struct PerRayData_radiance
{
	bool done;
	hiprandState * rngState;

	float pdfW;

	float3 nextPosition;
	float3 nextDirection;
	float3 flux;

	unsigned int photonIndex;
	int flag;
};

rtDeclareVariable(PerRayData_radiance, prdRadiance, rtPayload, );

// ray type 1
struct PerRayData_shadow
{
	bool hit;
};
rtDeclareVariable(PerRayData_shadow, prdShadow, rtPayload, );

///////////////////////////// Light Trace Closest Hit & Any Hit //////////////////////////

__device__ float russianProb(const float3 & throughput)
{
	return min(max(throughput.x, max(throughput.y, throughput.z)), 0.98f);
}

__device__ float pdfW2A(const float3 & n2, const float3 & v12)
{
	float3 nv12 = normalize(v12);
	return max(-dot(n2, nv12), 0.f) / dot(v12, v12);
}

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometryNormal, attribute geometryNormal, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtTextureSampler<float4, 2> lambertReflectanceTexture;
rtTextureSampler<float4, 2> phongReflectanceTexture;
rtTextureSampler<float4, 2> phongExponentTexture;
rtDeclareVariable(float4, lightIntensity, , );

RT_PROGRAM void rtMaterialClosestHit()
{
	float3 worldGeometryNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometryNormal));
	float3 ffNormal = faceforward(worldGeometryNormal, -ray.direction, worldGeometryNormal);

	// update position and normal
	float3 position = prdRadiance.nextPosition;
	float3 nextPosition = ray.origin + tHit * ray.direction;
	float3 nextNormal = ffNormal;

	// reject the result if normal is in the other direction or it's light source
	if (dot(geometryNormal, ray.direction) > 0.f || lightIntensity.x > 0.01f)
	{
		prdRadiance.done = true;
		return;
	}

	// fetch all texture information
	float3 lambertReflectance = make_float3(tex2D(lambertReflectanceTexture, texcoord.x, texcoord.y));
	float3 phongReflectance = make_float3(tex2D(phongReflectanceTexture, texcoord.x, texcoord.y));
	float phongExponent = tex2D(phongExponentTexture, texcoord.x, texcoord.y).x;

	const unsigned int index = prdRadiance.photonIndex;

	float3 direction;
	float pdfW;

	// sample next direction from previous direction
	float maxLambert = MaxColor(lambertReflectance);
	float maxPhong = MaxColor(phongReflectance);
	if (maxLambert + maxPhong <= 0.000001f)
	{
		prdRadiance.done = true;
		return;
	}

	photons[index].mFluxDir = -ray.direction;
	photons[index].mPosition = nextPosition;
	photons[index].mNormal = nextNormal;
	photons[index].mFlux = prdRadiance.flux;
	photons[index].mLambertReflectance = lambertReflectance;
	photons[index].mPhongReflectance = phongReflectance;
	photons[index].mPhongExponent = phongExponent;
	photons[index].mFlags = prdRadiance.flag;

	ASSERT(!isnan(prdRadiance.flux.x) && !isnan(prdRadiance.flux.y) && !isnan(prdRadiance.flux.z), "prdRadiance.flux(1) is nan");
	float pSelectLambert = maxLambert / (maxPhong + maxLambert);
	float chooseMaterial = min(hiprand_uniform(prdRadiance.rngState), 0.999999f);
	photons[index].mPSelectLambert = pSelectLambert;

	// russian roulette
	float russian = russianProb(prdRadiance.flux);
	prdRadiance.flux /= russian;
	prdRadiance.done = (hiprand_uniform(prdRadiance.rngState) >= russian);
	if (prdRadiance.done) { return; }

	if (chooseMaterial < pSelectLambert)
	{
		prdRadiance.flux *= LambertSample(&direction, &pdfW, -ray.direction, nextNormal, lambertReflectance, prdRadiance.rngState) / pSelectLambert;
		photons[index].mFlags = prdRadiance.flag | PhotonRecordFlag::LambertOnly;
	}
	else
	{
		prdRadiance.flux *= PhongSample(&direction, &pdfW, -ray.direction, geometryNormal, phongReflectance, phongExponent, prdRadiance.rngState) / (1.0f - pSelectLambert);
		photons[index].mFlags = prdRadiance.flag | PhotonRecordFlag::PhongOnly;
	}

	prdRadiance.nextPosition = nextPosition;
	prdRadiance.nextDirection = direction;
}

RT_PROGRAM void rtMaterialAnyHit()
{
	prdShadow.hit = true;
	rtTerminateRay();
}

///////////////////////////////////// LIGHT TRACING ////////////////////////////////

RT_PROGRAM void tracePhotons()
{
	unsigned int launchId = launchIndex.y * launchDimension.x + launchIndex.x;
	unsigned int pmIndex = launchId * numPhotonsPerLightPath;

	for (unsigned int i = 0;i < numPhotonsPerLightPath;i++)
	{
		photons[i + pmIndex].mFlags = 0;
	}

	hiprandState localState;
	hiprand_init(launchIndex.y * launchDimension.x + launchIndex.x, rngSeed, 0, &localState);

	// position and direction of first photon
	float3 position, normal;
	float pdf;
	float3 flux = LightSample(&position, &normal, &pdf, &localState);

	// sample outgoing direction from cosine weighted 
	float3 direction;
	float phongPdf;
	float3 att = PhongSample(&direction, &phongPdf, normal, normal, make_float3(1.0f), areaLightIntensity.w, &localState);

	RtPhotonRecord & photon = photons[pmIndex];
	photon.mPosition = position;
	photon.mNormal = normal;
	photon.mFlux = flux;
	photon.mFlags = PhotonRecordFlag::IsUsableVpl;
	photon.mPSelectLambert = 0.0f;

	photon.mLambertReflectance = make_float3(0.0f);
	photon.mPhongReflectance = make_float3(1.0f);
	photon.mPhongExponent = areaLightIntensity.w;
	photon.mFluxDir = normal;

	PerRayData_radiance prd;
	prd.rngState = &localState;
	prd.flux = flux * att;
	prd.done = false;
	prd.nextPosition = position;
	prd.nextDirection = direction;

	for (unsigned int i = 1;i < numPhotonsPerLightPath;i++)
	{
		Ray ray(prd.nextPosition, prd.nextDirection, 0, 0.0001f);

		prd.photonIndex = pmIndex + i;
		if (i != numPhotonsPerLightPath - 1)
		{
			prd.flag = PhotonRecordFlag::IsUsableVpl | PhotonRecordFlag::IsUsablePhoton;
		}
		else
		{
			prd.flag = PhotonRecordFlag::IsUsablePhoton;
		}
		rtTrace(topObject, ray, prd);
		if (prd.done) { break; }
	}
}

//////////////////////////////////// VPL SPLAT /////////////////////////////////////

__forceinline__ __device__ float BalanceHeuristic(const float pdfA, const float pdfB)
{
	return pdfA / (pdfA + pdfB);
}

__forceinline__ __device__ float MaxHeuristic(const float pdfA, const float pdfB)
{
	if (pdfA > pdfB)
	{
		return 1;
	}
	return 0;
}

__forceinline__ __device__ float PowerHeuristic2(const float pdfA, const float pdfB)
{
	float pdfA2 = pdfA * pdfA;
	float pdfB2 = pdfB * pdfB;
	return BalanceHeuristic(pdfA2, pdfB2);
}

__device__ float3 vplSplat(
	const float3 & wi10, // from shading point to eye
	const float3 & firstPosition, const float3 & firstNormal,
	const float3 & firstLambertReflectance, const float3 & firstPhongReflectance, const float firstPhongExponent,
	const RtPhotonRecord & photonRecord
)
{
	float3 v12 = photonRecord.mPosition - firstPosition;

	float unnormCos1 = max(dot(firstNormal, v12), 0.0f);
	float unnormCos2 = max(-dot(photonRecord.mNormal, v12), 0.0f);
	float unnormCos1Cos2 = unnormCos1 * unnormCos2;

	if (unnormCos1Cos2 <= 0.000f) { return make_float3(0.0f); }

	PerRayData_shadow prd;
	prd.hit = false;
	Ray ray(photonRecord.mPosition, -v12, 1, 0.0001, 1 - 0.0001);
	rtTrace(topObject, ray, prd);
	if (prd.hit) { return make_float3(0.0f); }

	float dist2 = dot(v12, v12);
	float dist = sqrtf(dist2);

	float3 wi12 = v12 / dist;
	float3 incomingDir = photonRecord.mFluxDir;

	float3 brdf2 = LambertEvalF(-wi12, incomingDir, photonRecord.mNormal) * photonRecord.mLambertReflectance
		+ PhongEvalF(-wi12, incomingDir, photonRecord.mNormal, photonRecord.mPhongExponent) * photonRecord.mPhongReflectance;

	float3 brdf1 = LambertEvalF(wi10, wi12, firstNormal) * firstLambertReflectance
		+ PhongEvalF(wi10, wi12, firstNormal, firstPhongExponent) * firstPhongReflectance;

	float g21 = unnormCos1Cos2 / (dist2 * dist2);

	if (misMode == 0)
	{
		return photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 1) // Balance Heuristic
	{
		float pdfDe = LambertPdfA(photonRecord.mNormal, firstNormal, -v12) * photonRecord.mPSelectLambert;
		pdfDe += PhongPdfA(photonRecord.mNormal, firstNormal, -v12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent) * (1.0f - photonRecord.mPSelectLambert);

		float weight = BalanceHeuristic(pdfMc, pdfDe);
		return weight * photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 2) // Max Heuristic
	{
		float pdfDe = LambertPdfA(photonRecord.mNormal, firstNormal, -v12) * photonRecord.mPSelectLambert;
		pdfDe += PhongPdfA(photonRecord.mNormal, firstNormal, -v12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent) * (1.0f - photonRecord.mPSelectLambert);

		float weight = MaxHeuristic(pdfMc, pdfDe);
		return weight * photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 3) // Power Heuristic
	{
		float pdfDe = LambertPdfA(photonRecord.mNormal, firstNormal, -v12) * photonRecord.mPSelectLambert;
		pdfDe += PhongPdfA(photonRecord.mNormal, firstNormal, -v12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent) * (1.0f - photonRecord.mPSelectLambert);

		float weight = PowerHeuristic2(pdfMc, pdfDe);
		return weight * photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 4) // KK weak singularities clamping
	{
		return photonRecord.mFlux * optix::fminf(g21, clampingValue) * brdf1 * brdf2;
	}
	else if (misMode == 5) // Local VPLs clamping
	{
		return photonRecord.mFlux * optix::fminf(g21 * brdf1 * brdf2, make_float3(clampingValue));
	}
}

RT_PROGRAM void splatColor()
{
	float2 screenUv = (make_float2(launchIndex) + make_float2(0.5)) / make_float2(launchDimension);
	float4 positionInfo = tex2D(deferredPositionTexture, screenUv.x, screenUv.y);
	float3 firstPosition = make_float3(positionInfo);
	float stencil = positionInfo.w;
	if (stencil == 0.0f) { return; }

	float3 firstNormal = make_float3(tex2D(deferredNormalTexture, screenUv.x, screenUv.y));
	float3 lambertReflectance = make_float3(tex2D(deferredDiffuseTexture, screenUv.x, screenUv.y));
	float4 phongInfo = tex2D(deferredPhongReflectanceTexture, screenUv.x, screenUv.y);

	float3 phongReflectance = make_float3(phongInfo);
	float phongExponent = phongInfo.w;

	float3 wi01 = normalize(cameraPosition - firstPosition); // from shading point to eye
															 //rtPrintf("%f %f %f", wi01.x, wi01.y, wi01.z);

	float3 result = make_float3(0.0f);

	unsigned numPhotons = numPhotonsPerLightPath * numVplLightPaths;

	for (int i = 0;i < numPhotons;i++)
	{
		if ((photons[i].mFlags & PhotonRecordFlag::IsUsableVpl) != 0)
		{
			result += vplSplat(wi01, firstPosition, firstNormal, lambertReflectance, phongReflectance, phongExponent, photons[i]);
		}
	}

	outputBuffer[launchIndex] = make_float4(result / (float) numVplLightPaths) + doAccumulate * outputBuffer[launchIndex];
}

// taken from Total Compendium pg. 19 (34)
__device__ float3 SquareToSolidAngle(const float sampleX, const float sampleY, const float halfAngleMax)
{
	const float phi = 2.0f * M_PIf * sampleX;
	const float z = 1.0f - sampleY * (1.0f - cosf(halfAngleMax));
	const float l = sqrtf(1.0f - z * z);
	const float cosphi = cosf(phi);
	const float sinphi = sinf(phi);
	return make_float3(cosphi * l, sinphi * l, z);
}

//////////////////////////////////////// VSL //////////////////////////////////////////////
// the following code is a translation from http://miloshasan.net/VirtualSphericalLights/vsl.fx

__device__ float3 sampleCone(float * misWeight,
							 const float3 & wi01,
							 const float3 & position,
							 const float3 & normal,
							 const float3 & lambertRefl,
							 const float3 & phongRefl,
							 const float phongExp,
							 const RtPhotonRecord & photonRecord,
							 const float halfCone,
							 const float solidAngle,
							 const float invSolidAngle,
							 const float3 & nd12,
							 hiprandState * rngState)
{
	float maxLambert = MaxColor(lambertRefl);
	float maxPhong = MaxColor(phongRefl);
	if (maxLambert + maxPhong <= 0.000001f) { return make_float3(0.0f); }

	float pSelectLambert = maxLambert / (maxPhong + maxLambert);
	float chooseMaterial = min(hiprand_uniform(rngState), 0.999999f);

	// sample outgoing direction from cosine weighted
	float3 wi12 = normalize(SquareToSolidAngle(hiprand_uniform(rngState), hiprand_uniform(rngState), halfCone));
	Onb onb(nd12);
	onb.inverse_transform(wi12);
	wi12 = normalize(wi12);

	const float cos1cos2 = fmaxf(dot(normal, wi12), 0.0f) * fmaxf(-dot(photonRecord.mNormal, wi12), 0.0f);
	if (cos1cos2 <= 0.000000001f) { return make_float3(0.0f); }

	float3 incomingDir = photonRecord.mFluxDir;

	float3 brdf2 = LambertEvalF(-wi12, incomingDir, photonRecord.mNormal) * photonRecord.mLambertReflectance
		+ PhongEvalF(-wi12, incomingDir, photonRecord.mNormal, photonRecord.mPhongExponent) * photonRecord.mPhongReflectance;

	float3 brdf1 = LambertEvalF(wi01, wi12, normal) * lambertRefl
		+ PhongEvalF(wi01, wi12, normal, phongExp) * phongRefl;

	float pdfCone = invSolidAngle;

	// compute pdfBrdf1
	float pdfBrdf1 = LambertPdfW(normal, wi12) * pSelectLambert +
		PhongPdfW(normal, wi12, wi01, phongRefl, phongExp) * (1.0f - pSelectLambert);

	// compute pdfBrdf2
	float pdfBrdf2 = LambertPdfW(photonRecord.mNormal, -wi12) * pSelectLambert +
		PhongPdfW(photonRecord.mNormal, -wi12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent);

	*misWeight = pdfCone / (pdfBrdf1 + pdfBrdf2 + pdfCone);

	return photonRecord.mFlux * vslInvPiRadius2 * cos1cos2 * brdf1 * brdf2 * solidAngle;
}

__device__ float3 sampleBrdf1(float * misWeight,
							  const float3 & wi01,
							  const float3 & position,
							  const float3 & normal,
							  const float3 & lambertRefl,
							  const float3 & phongRefl,
							  const float phongExp,
							  const RtPhotonRecord & photonRecord,
							  const float cosHalfCone,
							  const float invSolidAngle,
							  const float3 & nd12,
							  hiprandState * rngState)
{
	// sample brdf
	float3 wi12;
	float3 brdf1;
	float pdfW;
	{
		// sample next wi12 from previous wi12
		float maxLambert = MaxColor(lambertRefl);
		float maxPhong = MaxColor(phongRefl);
		if (maxLambert + maxPhong <= 0.000001f) { return make_float3(0.0f); }

		float pSelectLambert = maxLambert / (maxPhong + maxLambert);
		float chooseMaterial = min(hiprand_uniform(rngState), 0.999999f);

		if (chooseMaterial < pSelectLambert)
		{
			brdf1 = LambertSample(&wi12, &pdfW, wi01, normal, lambertRefl, rngState) / pSelectLambert;
		}
		else
		{
			brdf1 = PhongSample(&wi12, &pdfW, wi01, normal, phongRefl, phongExp, rngState) / (1.0f - pSelectLambert);
		}
	}


	if (dot(wi12, nd12) <= cosHalfCone)
	{
		return make_float3(0.0f);
	}

	const float cos1 = fmaxf(dot(normal, wi12), 0.0f);
	if (cos1 <= 0.000000001f) { return make_float3(0.0f); }

	const float cos2 = fmaxf(-dot(photonRecord.mNormal, wi12), 0.0f);

	float3 incomingDir = photonRecord.mFluxDir;

	float3 brdf2 = LambertEvalF(-wi12, incomingDir, photonRecord.mNormal) * photonRecord.mLambertReflectance
		+ PhongEvalF(-wi12, incomingDir, photonRecord.mNormal, photonRecord.mPhongExponent) * photonRecord.mPhongReflectance;

	{
		float maxLambert = MaxColor(lambertRefl);
		float maxPhong = MaxColor(phongRefl);
		if (maxLambert + maxPhong <= 0.000001f) { return make_float3(0.0f); }

		float pSelectLambert = maxLambert / (maxPhong + maxLambert);
		float chooseMaterial = min(hiprand_uniform(rngState), 0.999999f);

		float pdfCone = invSolidAngle;

		// compute pdfBrdf1
		float pdfBrdf1 = LambertPdfW(normal, wi12) * pSelectLambert +
			PhongPdfW(normal, wi12, wi01, phongRefl, phongExp) * (1.0f - pSelectLambert);

		// compute pdfBrdf2
		float pdfBrdf2 = LambertPdfW(photonRecord.mNormal, -wi12) * pSelectLambert +
			PhongPdfW(photonRecord.mNormal, -wi12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent);

		*misWeight = pdfBrdf1 / (pdfBrdf1 + pdfBrdf2 + pdfCone);
	}
	return photonRecord.mFlux * vslInvPiRadius2 * cos2 * brdf1 * brdf2;
}

__device__ float3 sampleBrdf2(float * misWeight,
							  const float3 & wi10,
							  const float3 & position,
							  const float3 & normal,
							  const float3 & lambertRefl,
							  const float3 & phongRefl,
							  const float phongExp,
							  const RtPhotonRecord & photonRecord,
							  const float cosHalfCone,
							  const float invSolidAngle,
							  const float3 & nd12,
							  hiprandState * rngState)
{
	float3 wi21;
	float3 brdf2;

	const float3 & incomingDir = photonRecord.mFluxDir;
	float pdfW;
	{
		// sample next wi12 from previous wi12
		float maxLambert = MaxColor(photonRecord.mLambertReflectance);
		float maxPhong = MaxColor(photonRecord.mPhongReflectance);
		if (maxLambert + maxPhong <= 0.000001f) { return make_float3(0.0f); }

		float pSelectLambert = maxLambert / (maxPhong + maxLambert);
		float chooseMaterial = min(hiprand_uniform(rngState), 0.999999f);

		if (chooseMaterial < pSelectLambert)
		{
			brdf2 = LambertSample(&wi21, &pdfW, incomingDir, photonRecord.mNormal, photonRecord.mLambertReflectance, rngState) / pSelectLambert;
		}
		else
		{
			brdf2 = PhongSample(&wi21, &pdfW, incomingDir, photonRecord.mNormal, photonRecord.mPhongReflectance, photonRecord.mPhongExponent, rngState) / (1.0f - pSelectLambert);
		}
	}

	if (-dot(wi21, nd12) <= cosHalfCone)
	{
		return make_float3(0.0f);
	}

	float3 brdf1 = LambertEvalF(wi10, -wi21, normal) * lambertRefl
		+ PhongEvalF(wi10, -wi21, normal, phongExp) * phongRefl;

	const float cos2 = fmaxf(dot(photonRecord.mNormal, wi21), 0.0f);
	if (cos2 <= 0.00000001f) { return make_float3(0.0f); }

	const float cos1 = fmaxf(-dot(normal, wi21), 0.0f);

	{
		float maxLambert = MaxColor(lambertRefl);
		float maxPhong = MaxColor(phongRefl);
		if (maxLambert + maxPhong <= 0.000001f) { return make_float3(0.0f); }

		float pSelectLambert = maxLambert / (maxPhong + maxLambert);
		float chooseMaterial = min(hiprand_uniform(rngState), 0.999999f);

		float pdfCone = invSolidAngle;

		// compute pdfBrdf1
		float pdfBrdf1 = LambertPdfW(normal, -wi21) * pSelectLambert +
			PhongPdfW(normal, -wi21, wi10, phongRefl, phongExp) * (1.0f - pSelectLambert);

		// compute pdfBrdf2
		float pdfBrdf2 = LambertPdfW(photonRecord.mNormal, wi21) * pSelectLambert +
			PhongPdfW(photonRecord.mNormal, wi21, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent);

		*misWeight = pdfBrdf2 / (pdfBrdf1 + pdfBrdf2 + pdfCone);
	}
	return photonRecord.mFlux * vslInvPiRadius2 * cos1 * brdf1 * brdf2;
}

__device__ float3 vslSplat(const float3 & wi10, // from shading point to eye
						   const float3 & firstPosition,
						   const float3 & firstNormal,
						   const float3 & firstLambertReflectance,
						   const float3 & firstPhongReflectance,
						   const float firstPhongExponent,
						   const RtPhotonRecord & photonRecord,
						   hiprandState * localState)
{
	float3 v12 = photonRecord.mPosition - firstPosition;
	float dist2 = dot(v12, v12);
	float dist = sqrtf(dist2);

	PerRayData_shadow prd;
	prd.hit = false;
	/// TODO:: the shadow ray bias should actually depends on length of ray.
	Ray ray(photonRecord.mPosition, -v12, 1, 0.0001, 1 - 0.0001); 
	rtTrace(topObject, ray, prd);
	if (prd.hit) { return make_float3(0.0f); }

	float3 nv12 = v12 / dist;

	const float cos1cos2 = fmaxf(dot(firstNormal, nv12), 0.0f) * fmaxf(-dot(photonRecord.mNormal, nv12), 0.0f);
	if (cos1cos2 <= 0.000000001f) { return make_float3(0.0f); }

	const float rdratio = vslRadius / dist;
	// asinf is sensitive and could produce nan if rdratio is too low
	const float halfCone = (rdratio >= 1.0) ? M_PIf / 2.0f : asinf(rdratio);// asinf(max(rdratio, 0.0005f));
	const float cosHalfCone = cosf(halfCone);
	const float solidAngle = M_PIf * 2.0f * (1.0f - cosHalfCone);
	const float invSolidAngle = 1.0f / solidAngle;

	float3 result = make_float3(0.0f);

	// compute num samples

	int numSamples = (int)(halfCone / M_PIf * 2.0f * 100.0f) + 1;

	for (int i = 0;i < numSamples;i++)
	{
		float coneWeight = 0.0f;
		float3 coneResult = sampleCone(&coneWeight,
									   wi10,
									   firstPosition,
									   firstNormal,
									   firstLambertReflectance,
									   firstPhongReflectance,
									   firstPhongExponent,
									   photonRecord,
									   halfCone,
									   solidAngle,
									   invSolidAngle,
									   nv12,
									   localState);

		float brdf1Weight = 0.0f;
		float3 brdf1Result = sampleBrdf1(&brdf1Weight,
										 wi10,
										 firstPosition,
										 firstNormal,
										 firstLambertReflectance,
										 firstPhongReflectance,
										 firstPhongExponent,
										 photonRecord,
										 cosHalfCone,
										 invSolidAngle,
										 nv12,
										 localState);

		float brdf2Weight = 0.0f;
		float3 brdf2Result = sampleBrdf2(&brdf2Weight,
										 wi10,
										 firstPosition,
										 firstNormal,
										 firstLambertReflectance,
										 firstPhongReflectance,
										 firstPhongExponent,
										 photonRecord,
										 cosHalfCone,
										 invSolidAngle,
										 nv12,
										 localState);

		//result += (coneWeight * coneResult + brdf1Weight * brdf1Result + brdf2Weight * brdf2Result);
		result += coneWeight * coneResult;
		result += brdf1Weight * brdf1Result;
		result += brdf2Weight * brdf2Result;
	}

	return result / (float)numSamples;
}

// VSL main program
RT_PROGRAM void splatSplotch()
{
	float2 screenUv = (make_float2(launchIndex) + make_float2(0.5)) / make_float2(launchDimension);
	float4 positionInfo = tex2D(deferredPositionTexture, screenUv.x, screenUv.y);
	float3 firstPosition = make_float3(positionInfo);
	//float stencil = positionInfo.w;
	//if (stencil == 0.0f) { return; }

	float3 firstNormal = make_float3(tex2D(deferredNormalTexture, screenUv.x, screenUv.y));
	float3 lambertReflectance = make_float3(tex2D(deferredDiffuseTexture, screenUv.x, screenUv.y));
	float4 phongInfo = tex2D(deferredPhongReflectanceTexture, screenUv.x, screenUv.y);

	float3 phongReflectance = make_float3(phongInfo);
	float phongExponent = phongInfo.w;

	float3 wi10 = normalize(cameraPosition - firstPosition); // from shading point to eye
															 //rtPrintf("%f %f %f", wi01.x, wi01.y, wi01.z);
	float3 result = make_float3(0.0f);

	unsigned numPhotons = numPhotonsPerLightPath * numVplLightPaths;

	hiprandState localState;
	hiprand_init(launchIndex.y * launchDimension.x + launchIndex.x, rngSeed, 0, &localState);

	for (int i = 0;i < numPhotons;i++)
	{
		if ((photons[i].mFlags & PhotonRecordFlag::IsUsableVpl) != 0)
		{
			result += vslSplat(wi10, firstPosition, firstNormal, lambertReflectance, phongReflectance, phongExponent, photons[i], &localState);
		}
	}

	outputBuffer[launchIndex] = make_float4(result / (float) numVplLightPaths) + doAccumulate * outputBuffer[launchIndex];
}
