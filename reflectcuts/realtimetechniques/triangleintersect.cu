#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_device.h>

using namespace optix;

rtBuffer<float3> vertexBuffer;
rtBuffer<int3> indexBuffer;
rtBuffer<float2> texcoordBuffer;

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometryNormal, attribute geometryNormal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void meshFineIntersect(int primIndex)
{
	int3 vertexIndex = indexBuffer[primIndex];

	float3 p0 = vertexBuffer[vertexIndex.x];
	float3 p1 = vertexBuffer[vertexIndex.y];
	float3 p2 = vertexBuffer[vertexIndex.z];

	float3 n;
	float t, beta, gamma;
	if (optix::intersect_triangle_branchless(ray, p0, p1, p2, n, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			geometryNormal = normalize(n);

			float2 t0 = texcoordBuffer[vertexIndex.x];
			float2 t1 = texcoordBuffer[vertexIndex.y];
			float2 t2 = texcoordBuffer[vertexIndex.z];
			texcoord = t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma);

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void meshIntersect(int primIndex)
{
	int3 vertexIndex = indexBuffer[primIndex];

	float3 p0 = vertexBuffer[vertexIndex.x];
	float3 p1 = vertexBuffer[vertexIndex.y];
	float3 p2 = vertexBuffer[vertexIndex.z];

	float3 n;
	float t, beta, gamma;
	if (optix::intersect_triangle_earlyexit(ray, p0, p1, p2, n, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			rtReportIntersection(0); // 0 = shadow material on host side
		}
	}
}

RT_PROGRAM void meshBound(int primIdx, float result[6])
{
	const int3 v_idx = indexBuffer[primIdx];

	const float3 v0 = vertexBuffer[v_idx.x];
	const float3 v1 = vertexBuffer[v_idx.y];
	const float3 v2 = vertexBuffer[v_idx.z];
	const float area = length(cross(v1 - v0, v2 - v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area))
	{
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	}
	else
	{
		aabb->invalidate();
	}
}