#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_device.h>

#include <hiprand/hiprand_kernel.h>

#include "rtmath.cuh"
#include "rtlightsource.cuh"
#include "rtmaterial.cuh"
#include "rtcomphoton/rtphotonrecord.h"

///// shared info /////

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDimension, rtLaunchDim, );

rtDeclareVariable(uint, rngSeed, , );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(uint, numLightPaths, , );
rtDeclareVariable(uint, numPhotonsPerLightPath, , );
rtDeclareVariable(uint, numVplLightPaths, , );

///// Light Trace info /////

rtBuffer<RtPhotonRecord, 1> photons;
rtBuffer<RtPhotonInfo, 1> photonInfo;

//rtDeclareVariable(uint, numMaxPhotonsPerLightPath, , );

///// VPL splat /////

rtBuffer<float4, 2> outputBuffer;

rtTextureSampler<float4, 2> deferredPositionTexture;
rtTextureSampler<float4, 2> deferredNormalTexture;
rtTextureSampler<float4, 2> deferredDiffuseTexture;
rtTextureSampler<float4, 2> deferredPhongReflectanceTexture;
rtTextureSampler<float4, 2> deferredPhongExpostureTexture;

rtDeclareVariable(float3, cameraPosition, , );
rtDeclareVariable(uint, doAccumulate, , );
rtDeclareVariable(float, pdfMc, , );
rtDeclareVariable(float, radius, , );
rtDeclareVariable(uint, misMode, , );
rtDeclareVariable(float, clampingValue, , );

rtDeclareVariable(float, vslRadius, , );
rtDeclareVariable(float, vslInvPiRadius2, , );

RT_PROGRAM void insertPhotons(
	const unsigned int pmIndex,
	const unsigned int numBounce,
	const float3 & position)
{
	RtPhotonRecord & rec = photons[pmIndex + numBounce];
	rec.mPosition = position;
}

RT_PROGRAM void exception()
{
	rtPrintExceptionDetails();
}

// ray type 0
struct PerRayData_radiance
{
	bool done;
	hiprandState * rngState;

	float pdfW;

	float3 nextPosition;
	float3 nextDirection;
	float3 flux;

	unsigned int photonIndex;
	int flag;
};

rtDeclareVariable(PerRayData_radiance, prdRadiance, rtPayload, );

// ray type 1
struct PerRayData_shadow
{
	bool hit;
};
rtDeclareVariable(PerRayData_shadow, prdShadow, rtPayload, );

///////////////////////////// Light Trace Closest Hit & Any Hit //////////////////////////

__device__ float russianProb(const float3 & throughput)
{
	return min(max(throughput.x, max(throughput.y, throughput.z)), 0.98f);
}

__device__ float pdfW2A(const float3 & n2, const float3 & v12)
{
	float3 nv12 = normalize(v12);
	return max(-dot(n2, nv12), 0.f) / dot(v12, v12);
}

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometryNormal, attribute geometryNormal, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtTextureSampler<float4, 2> lambertReflectanceTexture;
rtTextureSampler<float4, 2> phongReflectanceTexture;
rtTextureSampler<float4, 2> phongExponentTexture;
rtDeclareVariable(float4, lightIntensity, , );

RT_PROGRAM void rtMaterialClosestHit()
{
	float3 worldGeometryNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometryNormal));
	float3 ffNormal = faceforward(worldGeometryNormal, -ray.direction, worldGeometryNormal);

	// update position and normal
	float3 position = prdRadiance.nextPosition;
	float3 nextPosition = ray.origin + tHit * ray.direction;
	float3 nextNormal = ffNormal;

	// reject the result if normal is in the other direction or it's light source
	if (dot(geometryNormal, ray.direction) > 0.f || lightIntensity.x > 0.01f)
	{
		prdRadiance.done = true;
		return;
	}

	// fetch all texture information
	float3 lambertReflectance = make_float3(tex2D(lambertReflectanceTexture, texcoord.x, texcoord.y));
	float3 phongReflectance = make_float3(tex2D(phongReflectanceTexture, texcoord.x, texcoord.y));
	float phongExponent = tex2D(phongExponentTexture, texcoord.x, texcoord.y).x;

	const unsigned int index = prdRadiance.photonIndex;

	float3 direction;
	float pdfW;

	// sample next direction from previous direction
	float maxLambert = MaxColor(lambertReflectance);
	float maxPhong = MaxColor(phongReflectance);
	if (maxLambert + maxPhong <= 0.000001f)
	{
		prdRadiance.done = true;
		return;
	}

	photons[index].mFluxDir = -ray.direction;
	photons[index].mPosition = nextPosition;
	photons[index].mNormal = nextNormal;
	photons[index].mFlux = prdRadiance.flux;
	photons[index].mLambertReflectance = lambertReflectance;
	photons[index].mPhongReflectance = phongReflectance;
	photons[index].mPhongExponent = phongExponent;
	photons[index].mFlags = prdRadiance.flag;

	ASSERT(!isnan(prdRadiance.flux.x) && !isnan(prdRadiance.flux.y) && !isnan(prdRadiance.flux.z), "prdRadiance.flux(1) is nan");
	float pSelectLambert = maxLambert / (maxPhong + maxLambert);
	float chooseMaterial = min(hiprand_uniform(prdRadiance.rngState), 0.999999f);
	photons[index].mPSelectLambert = pSelectLambert;

	// russian roulette
	float russian = russianProb(prdRadiance.flux);
	prdRadiance.flux /= russian;
	prdRadiance.done = (hiprand_uniform(prdRadiance.rngState) >= russian);
	if (prdRadiance.done) { return; }

	if (chooseMaterial < pSelectLambert)
	{
		prdRadiance.flux *= LambertSample(&direction, &pdfW, -ray.direction, nextNormal, lambertReflectance, prdRadiance.rngState) / pSelectLambert;
		photons[index].mFlags = prdRadiance.flag | PhotonRecordFlag::LambertOnly;
	}
	else
	{
		prdRadiance.flux *= PhongSample(&direction, &pdfW, -ray.direction, geometryNormal, phongReflectance, phongExponent, prdRadiance.rngState) / (1.0f - pSelectLambert);
		photons[index].mFlags = prdRadiance.flag | PhotonRecordFlag::PhongOnly;
	}

	prdRadiance.nextPosition = nextPosition;
	prdRadiance.nextDirection = direction;
}

RT_PROGRAM void rtMaterialAnyHit()
{
	prdShadow.hit = true;
	rtTerminateRay();
}

///////////////////////////////////// LIGHT TRACING ////////////////////////////////

RT_PROGRAM void tracePhotons()
{
	unsigned int launchId = launchIndex.y * launchDimension.x + launchIndex.x;
	unsigned int pmIndex = launchId * numPhotonsPerLightPath;

	for (unsigned int i = 0;i < numPhotonsPerLightPath;i++)
	{
		photons[i + pmIndex].mFlags = 0;
	}

	hiprandState localState;
	hiprand_init(launchIndex.y * launchDimension.x + launchIndex.x, rngSeed, 0, &localState);

	// position and direction of first photon
	float3 position, normal;
	float pdf;
	float3 flux = LightSample(&position, &normal, &pdf, &localState);

	// sample outgoing direction from cosine weighted 
	float3 direction;
	float phongPdf;
	float3 att = PhongSample(&direction, &phongPdf, normal, normal, make_float3(1.0f), areaLightIntensity.w, &localState);

	RtPhotonRecord & photon = photons[pmIndex];
	photon.mPosition = position;
	photon.mNormal = normal;
	photon.mFlux = flux;
	photon.mFlags = PhotonRecordFlag::IsUsableVpl;
	photon.mPSelectLambert = 0.0f;

	photon.mLambertReflectance = make_float3(0.0f);
	photon.mPhongReflectance = make_float3(1.0f);
	photon.mPhongExponent = areaLightIntensity.w;
	photon.mFluxDir = normal;

	PerRayData_radiance prd;
	prd.rngState = &localState;
	prd.flux = flux * att;
	prd.done = false;
	prd.nextPosition = position;
	prd.nextDirection = direction;

	for (unsigned int i = 1;i < numPhotonsPerLightPath;i++)
	{
		Ray ray(prd.nextPosition, prd.nextDirection, 0, 0.0001f);

		prd.photonIndex = pmIndex + i;
		if (i != numPhotonsPerLightPath - 1)
		{
			prd.flag = PhotonRecordFlag::IsUsableVpl | PhotonRecordFlag::IsUsablePhoton;
		}
		else
		{
			prd.flag = PhotonRecordFlag::IsUsablePhoton;
		}
		rtTrace(topObject, ray, prd);
		if (prd.done) { break; }
	}
}

//////////////////////////////////// VPL SPLAT /////////////////////////////////////

__forceinline__ __device__ float BalanceHeuristic(const float pdfA, const float pdfB)
{
	return pdfA / (pdfA + pdfB);
}

__forceinline__ __device__ float MaxHeuristic(const float pdfA, const float pdfB)
{
	if (pdfA > pdfB)
	{
		return 1;
	}
	return 0;
}

__forceinline__ __device__ float PowerHeuristic2(const float pdfA, const float pdfB)
{
	float pdfA2 = pdfA * pdfA;
	float pdfB2 = pdfB * pdfB;
	return BalanceHeuristic(pdfA2, pdfB2);
}

__device__ float3 vplSplat(
	const float3 & wi10, // from shading point to eye
	const float3 & firstPosition, const float3 & firstNormal,
	const float3 & firstLambertReflectance, const float3 & firstPhongReflectance, const float firstPhongExponent,
	const RtPhotonRecord & photonRecord
)
{
	float3 v12 = photonRecord.mPosition - firstPosition;

	float unnormCos1 = max(dot(firstNormal, v12), 0.0f);
	float unnormCos2 = max(-dot(photonRecord.mNormal, v12), 0.0f);
	float unnormCos1Cos2 = unnormCos1 * unnormCos2;

	if (unnormCos1Cos2 <= 0.000f) { return make_float3(0.0f); }

	PerRayData_shadow prd;
	prd.hit = false;
	Ray ray(photonRecord.mPosition, -v12, 1, 0.0001, 1 - 0.0001);
	rtTrace(topObject, ray, prd);
	if (prd.hit) { return make_float3(0.0f); }

	float dist2 = dot(v12, v12);
	float dist = sqrtf(dist2);

	float3 wi12 = v12 / dist;
	float3 incomingDir = photonRecord.mFluxDir;

	float3 brdf2 = LambertEvalF(-wi12, incomingDir, photonRecord.mNormal) * photonRecord.mLambertReflectance
		+ PhongEvalF(-wi12, incomingDir, photonRecord.mNormal, photonRecord.mPhongExponent) * photonRecord.mPhongReflectance;

	float3 brdf1 = LambertEvalF(wi10, wi12, firstNormal) * firstLambertReflectance
		+ PhongEvalF(wi10, wi12, firstNormal, firstPhongExponent) * firstPhongReflectance;

	float g21 = unnormCos1Cos2 / (dist2 * dist2);

	if (misMode == 0)
	{
		return photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 1) // Balance Heuristic
	{
		float pdfDe = LambertPdfA(photonRecord.mNormal, firstNormal, -v12) * photonRecord.mPSelectLambert;
		pdfDe += PhongPdfA(photonRecord.mNormal, firstNormal, -v12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent) * (1.0f - photonRecord.mPSelectLambert);

		float weight = BalanceHeuristic(pdfMc, pdfDe);
		return weight * photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 2) // Max Heuristic
	{
		float pdfDe = LambertPdfA(photonRecord.mNormal, firstNormal, -v12) * photonRecord.mPSelectLambert;
		pdfDe += PhongPdfA(photonRecord.mNormal, firstNormal, -v12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent) * (1.0f - photonRecord.mPSelectLambert);

		float weight = MaxHeuristic(pdfMc, pdfDe);
		return weight * photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 3) // Power Heuristic
	{
		float pdfDe = LambertPdfA(photonRecord.mNormal, firstNormal, -v12) * photonRecord.mPSelectLambert;
		pdfDe += PhongPdfA(photonRecord.mNormal, firstNormal, -v12, photonRecord.mFluxDir, photonRecord.mPhongReflectance, photonRecord.mPhongExponent) * (1.0f - photonRecord.mPSelectLambert);

		float weight = PowerHeuristic2(pdfMc, pdfDe);
		return weight * photonRecord.mFlux * brdf1 * brdf2 * g21;
	}
	else if (misMode == 4) // KK weak singularities clamping
	{
		return photonRecord.mFlux * optix::fminf(g21, clampingValue) * brdf1 * brdf2;
	}
	else if (misMode == 5) // Local VPLs clamping
	{
		return photonRecord.mFlux * optix::fminf(g21 * brdf1 * brdf2, make_float3(clampingValue));
	}
}

RT_PROGRAM void splatColor()
{
	float2 screenUv = (make_float2(launchIndex) + make_float2(0.5)) / make_float2(launchDimension);
	float4 positionInfo = tex2D(deferredPositionTexture, screenUv.x, screenUv.y);
	float3 firstPosition = make_float3(positionInfo);
	float stencil = positionInfo.w;
	if (stencil == 0.0f) { return; }

	float3 firstNormal = make_float3(tex2D(deferredNormalTexture, screenUv.x, screenUv.y));
	float3 lambertReflectance = make_float3(tex2D(deferredDiffuseTexture, screenUv.x, screenUv.y));
	float4 phongInfo = tex2D(deferredPhongReflectanceTexture, screenUv.x, screenUv.y);

	float3 phongReflectance = make_float3(phongInfo);
	float phongExponent = phongInfo.w;

	float3 wi01 = normalize(cameraPosition - firstPosition); // from shading point to eye
															 //rtPrintf("%f %f %f", wi01.x, wi01.y, wi01.z);

	float3 result = make_float3(0.0f);

	hiprandState localState;
	hiprand_init(launchIndex.y * launchDimension.x + launchIndex.x, rngSeed, 0, &localState);

	unsigned int lightPathOffset = unsigned int(min(hiprand_uniform(&localState), 0.999999f) * numLightPaths);

	for (int i = 0;i < numVplLightPaths;i++)
	{
		unsigned int lightPathId = (i + lightPathOffset) % numLightPaths;
		unsigned int lightVertexOffset = lightPathId * numPhotonsPerLightPath;
		for (int j = 0;j < numPhotonsPerLightPath;j++)
		{
			if ((photons[lightVertexOffset + j].mFlags & PhotonRecordFlag::IsUsableVpl) != 0)
			{
				result += vplSplat(wi01, firstPosition, firstNormal, lambertReflectance, phongReflectance, phongExponent, photons[lightVertexOffset + j]);
			}
		}
	}

	outputBuffer[launchIndex] = make_float4(result / (float) numVplLightPaths) + doAccumulate * outputBuffer[launchIndex];
}
