#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_device.h>

#include <hiprand/hiprand_kernel.h>

#include "rtmath.cuh"
#include "rtlightsource.cuh"
#include "rtmaterial.cuh"

using namespace optix;

// path tracing stuffs

rtBuffer<float4, 2> outputBuffer;

rtTextureSampler<float4, 2> deferredPositionTexture;
rtTextureSampler<float4, 2> deferredNormalTexture;
rtTextureSampler<float4, 2> deferredDiffuseTexture;
rtTextureSampler<float4, 2> deferredPhongReflectanceTexture;

rtDeclareVariable(float3, cameraPosition, , );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(uint, doAccumulate, , );

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDimension, rtLaunchDim, );

rtDeclareVariable(uint, maxBounces, , );
rtDeclareVariable(uint, rngSeed, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

static __device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4(static_cast<unsigned char>(c.z * 255.99f),  /* B */
		static_cast<unsigned char>(c.y*255.99f),  /* G */
		static_cast<unsigned char>(c.x*255.99f),  /* R */
		255u);                                                 /* A */
}

__device__ float3 generatePointInSphere(const float2 & sample)
{
	const float z = 1.0f - 2.0f * sample.y;
	const float r = sqrt(sample.y * (1.0f - sample.y));
	const float phi = 2.0f * M_PIf * sample.x; // phi = [0, 2pi)
	const float cosphi = cos(phi);
	const float sinphi = sin(phi);
	return make_float3(2.0f * cosphi * r, 2.0f * sinphi * r, z);
}

__device__ float russianProb(const float3 & throughput)
{
	return max(max(throughput.x, 0.98f), max(throughput.y, throughput.z));
}

RT_PROGRAM void exception()
{
	rtPrintExceptionDetails();
}

// ray type 0
struct PerRayData_radiance
{
	bool done;
	bool hit;
	hiprandState * rngState;

	float brdfPdfW;
	float3 result;
	float3 position;
	float3 geometryNormal;
	float3 direction;
	float3 attenuation;
};
rtDeclareVariable(PerRayData_radiance, prdRadiance, rtPayload, );

// ray type 1
struct PerRayData_shadow
{
	bool hit;
};
rtDeclareVariable(PerRayData_shadow, prdShadow, rtPayload, );

///////////////////////////// MIS Stuffs /////////////////////////

__device__ float MisWeight(const float pdf1, const float pdf2)
{
	return pdf1 / (pdf1 + pdf2);
}

__device__ float pdfW2A(const float3 & n2, const float3 & v12)
{
	float3 nv12 = normalize(v12);
	return max(-dot(n2, nv12), 0.f) / dot(v12, v12);
}

///////////////////////// RT MATERIAL ///////////////////////////

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometryNormal, attribute geometryNormal, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtTextureSampler<float4, 2> lambertReflectanceTexture;
rtTextureSampler<float4, 2> phongReflectanceTexture;
rtTextureSampler<float4, 2> phongExponentTexture;
rtDeclareVariable(float4, lightIntensity, , );

//#define FAVOR_LIGHT_SAMPLE
//#define FAVOR_BSDF_SAMPLE

RT_PROGRAM void rtMaterialClosestHit()
{
	ASSERT(!isnan(prdRadiance.attenuation.x) && !isnan(prdRadiance.attenuation.y) && !isnan(prdRadiance.attenuation.z), "prdRadiance.atteanuation(1) is nan");
	//prdRadiance.hit = true;

	float3 worldGeometryNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometryNormal));
	float3 ffNormal = faceforward(worldGeometryNormal, -ray.direction, worldGeometryNormal);

	// update position and normal
	float3 nextPosition = ray.origin + tHit * ray.direction;
	float3 nextNormal = ffNormal;

	// reject the result if normal is in the other direction
	if (dot(geometryNormal, ray.direction) > 0.f)
	{
		prdRadiance.result = make_float3(0.0f);
		prdRadiance.done = true;
		return;
	}

	// check if it hit the light source
	if (lightIntensity.x > 0.01f)
	{
		// compute mis weight
		float brdfPdfA = (prdRadiance.brdfPdfW * pdfW2A(ffNormal, nextPosition - prdRadiance.position));
		float lightPdfA = LightPdfA();
		float weight = MisWeight(brdfPdfA, lightPdfA);
		#ifdef FAVOR_LIGHT_SAMPLE
			weight = 0.0f;
		#elif defined(FAVOR_BSDF_SAMPLE)
			weight = 1.0f;
		#endif
		prdRadiance.result = weight * prdRadiance.attenuation *
			PhongEvalF(geometryNormal, normalize(prdRadiance.position - nextPosition), geometryNormal, lightIntensity.w) * make_float3(lightIntensity);
		prdRadiance.done = true;
		return;
	}

	// this is last bounce. don't do next event estimation
	if (prdRadiance.done) { return; }

	float lightPdf;
	float3 lightPosition, lightNormal;
	float3 lightValue = LightSample(&lightPosition, &lightNormal, &lightPdf, prdRadiance.rngState);

	float3 toLight = lightPosition - nextPosition;
	float3 toLightNorm = normalize(toLight);

	Ray ray(lightPosition, -toLight, 1, 0.00001, 0.99999);
	PerRayData_shadow prd2;
	prd2.hit = false;
	rtTrace(topObject, ray, prd2);

	float3 lambertReflectance = make_float3(tex2D(lambertReflectanceTexture, texcoord.x, texcoord.y));
	float3 phongReflectance = make_float3(tex2D(phongReflectanceTexture, texcoord.x, texcoord.y));
	float phongExponent = tex2D(phongExponentTexture, texcoord.x, texcoord.y).x;

	// check bad color (for reduce bracnching)
	float maxLambert = MaxColor(lambertReflectance);
	float maxPhong = MaxColor(phongReflectance);
	prdRadiance.done = (maxLambert + maxPhong <= 0.000001f);
	if (prdRadiance.done) { return; }

	float pSelectLambert = maxLambert / (maxPhong + maxLambert);
	float chooseMaterial = min(hiprand_uniform(prdRadiance.rngState), 0.999999f);

	if (chooseMaterial < pSelectLambert)
	{
		ASSERT(1.0f >= pSelectLambert && pSelectLambert > 0.0f, "pSelectLambert is not in (0, 1]");

		if (!prd2.hit)
		{
			// next event estimation
			float brdfPdf = LambertPdfA(ffNormal, lightNormal, toLight);
			float weight = MisWeight(lightPdf, brdfPdf);
			#ifdef FAVOR_LIGHT_SAMPLE
				weight = 1.0f;
			#elif defined(FAVOR_BSDF_SAMPLE)
				weight = 0.0f;
			#endif
			prdRadiance.result = weight * lightValue * LambertEval(toLightNorm, normalize(prdRadiance.position - nextPosition), ffNormal, lambertReflectance) * GeometryTerm(ffNormal, lightNormal, toLight) * prdRadiance.attenuation / pSelectLambert
				* PhongEvalF(lightNormal, -toLightNorm, lightNormal, areaLightIntensity.w); // light source material
		}

		// sample outgoing direction
		prdRadiance.attenuation *= LambertSample(&prdRadiance.direction, &prdRadiance.brdfPdfW, normalize(prdRadiance.position - nextPosition), geometryNormal, lambertReflectance, prdRadiance.rngState) / pSelectLambert;
	}
	else
	{
		ASSERT(1.0f > pSelectLambert && pSelectLambert >= 0.0f, "pSelectLambert is not in [0, 1)");

		if (!prd2.hit)
		{
			// next event estimation
			float brdfPdf = PhongPdfA(ffNormal, lightNormal, toLight, normalize(prdRadiance.position - nextPosition), phongReflectance, phongExponent);
			float weight = MisWeight(lightPdf, brdfPdf);
			#ifdef FAVOR_LIGHT_SAMPLE
				weight = 1.0f;
			#elif defined(FAVOR_BSDF_SAMPLE)
				weight = 0.0f;
			#endif
			prdRadiance.result = weight * lightValue * PhongEval(toLightNorm, normalize(prdRadiance.position - nextPosition), ffNormal, phongReflectance, phongExponent) * GeometryTerm(ffNormal, lightNormal, toLight) * prdRadiance.attenuation / (1.0f - pSelectLambert)
				* PhongEvalF(lightNormal, -toLightNorm, lightNormal, areaLightIntensity.w); // light source material
		}
		prdRadiance.attenuation *= PhongSample(&prdRadiance.direction, &prdRadiance.brdfPdfW, normalize(prdRadiance.position - nextPosition), geometryNormal, phongReflectance, phongExponent, prdRadiance.rngState) / (1.0f - pSelectLambert);
	}

	float russian = russianProb(prdRadiance.attenuation);
	ASSERT(russian > 0.0f, "russian roulette prob is <= 0.0");
	prdRadiance.done = (hiprand_uniform(prdRadiance.rngState) >= russian);
	if (prdRadiance.done) { return; }

	prdRadiance.position = nextPosition;
	prdRadiance.attenuation /= russian;

	ASSERT(!isnan(prdRadiance.attenuation.x) && !isnan(prdRadiance.attenuation.y) && !isnan(prdRadiance.attenuation.z), "prdRadiance.atteanuation(2) is nan");
}

/////////////////////////// ANY HIT ////////////////////////////

RT_PROGRAM void rtMaterialAnyHit()
{
	prdShadow.hit = true;
	rtTerminateRay();
}

/////////////////////////////////////////////////////////////////

__device__ float3 pathTraceSimple(
	const float3 & cameraPos,
	const float3 & firstPosition,
	const float3 & firstNormal,
	const float3 & firstLambertReflectance,
	const float3 & firstPhongReflectance,
	const float & firstPhongExponent,
	hiprandState * rngState)
{
	float3 cameraVec = normalize(firstPosition - cameraPos);
	float3 result = make_float3(0.0f);

	PerRayData_radiance prd;
	prd.rngState = rngState;

	const unsigned int numSamples = 1;
	const float invNumSamples = 1.f / (float)numSamples;

	float3 position = firstPosition;
	float3 normal = firstNormal;

	for (int k = 0;k < numSamples;k++)
	{
		prd.position = firstPosition;
		prd.geometryNormal = firstNormal;
		prd.attenuation = make_float3(1.0);

		// first bounce
		{
			// sample light source
			float lightPdf;
			float3 lightPosition, lightNormal;
			float3 lightValue = LightSample(&lightPosition, &lightNormal, &lightPdf, rngState);

			float3 toLight = lightPosition - position;
			float3 toLightNorm = normalize(toLight);

			Ray ray(lightPosition, -toLight, 1, 0.0001f, 1.0f - 0.0001f);
			PerRayData_shadow prd2;
			prd2.hit = false;
			rtTrace(topObject, ray, prd2);

			// select material
			float maxLambert = MaxColor(firstLambertReflectance);
			float maxPhong = MaxColor(firstPhongReflectance);

			float pSelectLambert = maxLambert / (maxPhong + maxLambert);

			if (maxLambert + maxPhong <= 0.000001f) { return make_float3(0.0f); }
			float chooseMaterial = min(hiprand_uniform(prd.rngState), 0.999999f);
			if (chooseMaterial < pSelectLambert)
			{
				ASSERT(1.0f >= pSelectLambert && pSelectLambert > 0.0f, "pSelectLambert(a) is not in (0, 1]");

				if (!prd2.hit)
				{
					// compute mis weight
					float brdfPdf = LambertPdfA(normal, lightNormal, toLight);
					float weight = MisWeight(lightPdf, brdfPdf);
					#ifdef FAVOR_LIGHT_SAMPLE
						weight = 1.0f;
					#elif defined(FAVOR_BSDF_SAMPLE)
						weight = 0.0f;
					#endif
					result += weight * lightValue * LambertEval(-cameraVec, toLightNorm, normal, firstLambertReflectance) * GeometryTerm(normal, lightNormal, toLight) / pSelectLambert
						* PhongEvalF(lightNormal, -toLightNorm, lightNormal, areaLightIntensity.w); // light source material
				}

				prd.attenuation *= LambertSample(&prd.direction, &prd.brdfPdfW, -cameraVec, normal, firstLambertReflectance, prd.rngState) / pSelectLambert;
			}
			else
			{
				ASSERT(1.0f > pSelectLambert && pSelectLambert >= 0.0f, "pSelectLambert(a) is not in [0, 1)");

				if (!prd2.hit)
				{
					// compute mis weight
					float brdfPdf = PhongPdfA(normal, lightNormal, toLight, -cameraVec, firstPhongReflectance, firstPhongExponent);
					float weight = MisWeight(lightPdf, brdfPdf);
					#ifdef FAVOR_LIGHT_SAMPLE
						weight = 1.0f;
					#elif defined(FAVOR_BSDF_SAMPLE)
						weight = 0.0f;
					#endif
					result += weight * lightValue * PhongEval(-cameraVec, toLightNorm, normal, firstPhongReflectance, firstPhongExponent) * GeometryTerm(normal, lightNormal, toLight) / (1.0f - pSelectLambert)
						* PhongEvalF(lightNormal, -toLightNorm, lightNormal, areaLightIntensity.w); // light source material
				}

				prd.attenuation *= PhongSample(&prd.direction, &prd.brdfPdfW, -cameraVec, normal, firstPhongReflectance, firstPhongExponent, prd.rngState) / (1.0f - pSelectLambert);
			}
			ASSERT(!isnan(prd.attenuation.x) && !isnan(prd.attenuation.y) && !isnan(prd.attenuation.z), "prd.atteanuation(a) is nan");
		}

		for (size_t i = 0;i < maxBounces;i++)
		{
			prd.done = (i == maxBounces - 1);
			prd.result = make_float3(0.f);

			Ray ray(prd.position, prd.direction, 0, 0.00001);
			rtTrace(topObject, ray, prd);

			result += prd.result;

			if (prd.done) { break; }
		}
	}

	return result * invNumSamples;
}

RT_PROGRAM void splatColor()
{
	float2 screenUv = (make_float2(launchIndex) + make_float2(0.5)) / make_float2(launchDimension);
	float4 positionInfo = tex2D(deferredPositionTexture, screenUv.x, screenUv.y);
	float3 firstPosition = make_float3(positionInfo);
	float stencil = positionInfo.w;
	if (stencil == 0.0f) { return; }

	float3 firstNormal = make_float3(tex2D(deferredNormalTexture, screenUv.x, screenUv.y));
	float3 lambertReflectance = make_float3(tex2D(deferredDiffuseTexture, screenUv.x, screenUv.y));
	float4 phongInfo = tex2D(deferredPhongReflectanceTexture, screenUv.x, screenUv.y);
	float3 phongReflectance = make_float3(phongInfo);
	float phongExponent = phongInfo.w;

	hiprandState localState;
	hiprand_init(launchIndex.y * launchDimension.x + launchIndex.x, rngSeed, 0, &localState);

	float3 result = pathTraceSimple(cameraPosition, firstPosition, firstNormal, lambertReflectance, phongReflectance, phongExponent, &localState);
	ASSERT(!isnan(result.x) && !isnan(result.y) && !isnan(result.z), "result is nan");
	if (doAccumulate == 1)
	{
		outputBuffer[launchIndex] += make_float4(result);
	}
	else
	{
		outputBuffer[launchIndex] = make_float4(result);
	}
}
